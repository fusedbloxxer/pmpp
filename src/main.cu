
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vecAddKernel(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float *A_h, float *B_h, float *C_h, int n)
{
    float *A_d = nullptr, *B_d = nullptr, *C_d = nullptr;
    size_t bytes = n * sizeof(*A_d);

    // Allocate memory on device
    hipMalloc(&A_d, bytes);
    hipMalloc(&B_d, bytes);
    hipMalloc(&C_d, bytes);

    // Host -> Device
    hipMemcpy(A_d, A_h, bytes, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, bytes, hipMemcpyHostToDevice);

    // Launch kernel
    vecAddKernel<<<ceil(n / 256.0), 256>>>(A_d, B_d, C_d, n);

    // Device -> Host
    hipMemcpy(C_h, C_d, bytes, hipMemcpyDeviceToHost);

    // Free memory on device
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return;
}

int main()
{
    float A[5] = {1, 2, 3, 4, 5};
    float B[5] = {9, 8, 6, 6, 5};
    float C[5] = {0};

    vecAdd(A, B, C, sizeof(A) / sizeof(*A));

    for (int i = 0; i != 5; i++)
    {
        std::cout << C[i] << " " << std::endl;
    }

    return 0;
}
